#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<stdlib.h>
#include<iomanip>
#include<stdint.h>
#include<inttypes.h>
#include<string.h>
#include<vector>
#include<fstream>
#include"header.h"
const unsigned int num_bins_a = 1024;	//dump a 1445
const unsigned int num_bins_b = 1024;	//dump a 1445
const unsigned int threads_side = 31; 	// 32x32=1024 masssimo di thread in una block
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#define gpuErrchk(ans) {gpuAssert( (ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if(code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	}
}


void coord(float *vec_x, float *vec_y, int size_xy,  std::vector<event> vec_eventi, int num_pixel, float dimensioni_tracker, float dist_interazione, float dist_piani)
{
	for(int i=0; i<size_xy; i++)
		{
			vec_x[i] = ((dimensioni_tracker*vec_eventi[i].channel_ID())/num_pixel)-(dimensioni_tracker/2);
			vec_y[i] = dist_interazione + dist_piani*(vec_eventi[i].layer()-1);
			//std::cout << "X: " << vec_x[i] << "\ty: " << vec_y[i] << std::endl;
		}
}


void full_header(std::ifstream &data)
{
	// LEGGO PRIMA PAROLA, RITORNO ERRORE SE QUESTA NON E' "BABACACA"
	uint32_t word;
	data.read((char*) &word, sizeof(word));
	//std::cout << "0x" << std::hex << std::setw(8) << std::setfill('0') << word << std::endl;
	uint32_t w = uint32_t(word);
	if(w==0xbabacaca){std::cout << "Checkword Full Header Corretta" << std::endl;}
	else
	{
		std::cerr << "Checkword di inizio file non corrisponde! " ;
	}
	int size_full_header;
	data.read((char*) &size_full_header, sizeof(size_full_header));
	//size_full_header=int(word2);
	//std::cout << "0x" << std::hex << std::setw(8) << std::setfill('0') << word2 << std::endl;
	std::cout << "Dimensione Full Header: " << size_full_header << std::endl;
	uint32_t* buf = new uint32_t[size_full_header-2];
	for(int i=0; i<(size_full_header-2); i++)
	{
		data.read((char*) &buf[i], 4);
	}
	std::cout << "Data Format Versione: " << "0x" << std::hex << std::setw(8) << std::setfill('0') << buf[0] << std::endl;
	std::cout << "Identificazione Detector: " << "0x" << std::hex << std::setw(8) << std::setfill('0') << buf[1] << std::endl;
}

void fragment(std::ifstream &data, float dimensioni_tracker, int num_pixel, std::vector<event> &vec_eventi)
{
	uint32_t event_ID;
	data.read((char*) &event_ID, sizeof(event_ID));
	//if(event_ID==0x00ff00ff){return;}
	event_ID = int(event_ID);
	std::cout << "\nEvento_numero: " << std::dec << event_ID << std::endl;
	while(true)
	{
		uint32_t new_word;
		data.read((char*) &new_word, sizeof(new_word));
		// SE LA PAROLA CORRISPONDE AL TRAILER DI FINE EVENTO RITORNO 
		if(new_word==0x00ff00ff){ break;}
		event hit(new_word, event_ID);
		vec_eventi.push_back(hit);
	}
std::cout<< "Dimensione payload: " << vec_eventi.size() << std::endl;
}

__global__ void hough_gpu(matrix d_m, float* d_vec_bin, float* d_vec_x, float* d_vec_y, int size_xy)
{

	int index = blockIdx.x*blockDim.x + threadIdx.x;
	for(int k=0; k<size_xy; k++)
	{
		float elm = (d_vec_x[k]-d_vec_bin[index])/d_vec_y[k];
		d_m.e[k*d_m.w + index] = elm;
	}
} 


__global__ void hough_gpu_new(matrix d_m, float* d_vec_bin, float* d_vec_x, float* d_vec_y, int size_xy)
{
	int block_id = blockIdx.x;
	int index = threadIdx.x;
	float elm = (d_vec_x[block_id]-d_vec_bin[index])/d_vec_y[block_id];
	d_m.e[block_id*d_m.w + index] = elm;

} 

void hough_cpu(matrix &m, float* vettore_di_bin, float *vec_x, float *vec_y)
{
	for(int k=0; k<m.h; k++) // i indicatore della riga
		for(int j=0; j<m.w; j++) // j indicarore colonna
		{
			if(vec_x[k]==0){;}
			else m.e[k*m.w + j] = (vec_x[k]-vettore_di_bin[j])/vec_y[k];
			//std::cout << m.e[k*m.w + j] << " boh " ;
			//std:: cout << (vec_x[k]-vettore_di_bin[j])/vec_y[k] << " ";
		}
}



__global__ void ricerca_max_gpu(unsigned int *d_histo, float* d_vec_bin, float* d_vec_coeff, int event_ID, int threshold)
{
	int bin_a=threadIdx.x+blockIdx.x*blockDim.x;
	__shared__ int array_max[num_bins_a];
	int location=1;
	__shared__ int loc[num_bins_a];
	// cerco massimo lungo colonna bin offset -> Salvo valore massimo in array_max e bin coef. ang. in loc[]
	for(int c=0; c<num_bins_b; c++)
	{
		if(d_histo[bin_a*num_bins_b+c] > d_histo[bin_a*num_bins_b+location])
		{
			location=c;
		}
		loc[bin_a]=location;
		array_max[bin_a] = d_histo[bin_a*num_bins_b+location];
	}
	__syncthreads();

	if(bin_a>1 and bin_a<num_bins_a-1 and array_max[bin_a]>threshold and array_max[bin_a-1]<array_max[bin_a] and array_max[bin_a]>array_max[bin_a+1]){

                printf("Evento %d \t Massimo %d \tCoeffAng %.4f \toffset %.4f\n", event_ID, array_max[bin_a], d_vec_coeff[bin_a], d_vec_bin[loc[bin_a]]);
                }

}
__global__ void ricerca_max_gpu_locale2(unsigned int *d_histo, float *d_vec_bin, float* d_vec_coeff, int event_ID, int threshold)
{
	int bin_b=threadIdx.x + blockIdx.x*blockDim.x;
	int bin_a=threadIdx.y + blockIdx.y*blockDim.y;
	unsigned int channelID = bin_a*num_bins_b + bin_b;

	unsigned int dx = bin_a*num_bins_b+(bin_b+1);
	unsigned int sx = bin_a*num_bins_b+(bin_b-1);
	unsigned int up = (bin_a+1)*num_bins_b+bin_b;
	unsigned int down = (bin_a-1)*num_bins_b+bin_b;
	
	unsigned int dx2 = bin_a*num_bins_b+(bin_b+2);
	unsigned int sx2 = bin_a*num_bins_b+(bin_b-2);
	unsigned int up2 = (bin_a+2)*num_bins_b+bin_b;
	unsigned int down2 = (bin_a-2)*num_bins_b+bin_b;
	
	unsigned int ud = (bin_a+1)*num_bins_b+(bin_b+1);
	unsigned int us = (bin_a+1)*num_bins_b+(bin_b-1);
	unsigned int dd = (bin_a-1)*num_bins_b+(bin_b+1);
	unsigned int ds = (bin_a-1)*num_bins_b+(bin_b-1);
	// CASO GENERALE, VENGONO ESCLUSE LE THREAD AI BORDI DELLA BLOCK
	if(threadIdx.x>1 and threadIdx.x<threads_side-2 and threadIdx.y>1 and threadIdx.y<threads_side-2)
	{
		if(d_histo[channelID]>threshold and d_histo[channelID]>=d_histo[up] and d_histo[channelID]>=d_histo[down] and d_histo[channelID]>=d_histo[dx] and d_histo[channelID]>=d_histo[sx] and d_histo[channelID]>d_histo[up2] and d_histo[channelID]>d_histo[down2] and d_histo[channelID]>d_histo[dx2] and d_histo[channelID]>d_histo[sx2])
			{
				if(d_histo[channelID]>d_histo[ud] and d_histo[channelID]>d_histo[us] and d_histo[channelID]>d_histo[dd] and d_histo[channelID]>d_histo[ds])
					printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_a], d_vec_bin[bin_b]);
			}
	}
	
	// CONSIDERO LE THREAD LUNGO I BORDI DELLA BLOCK
	else if(threadIdx.x == 0 and threadIdx.y>1 and threadIdx.y<threads_side-2)
	{
		if(d_histo[channelID]>threshold and d_histo[channelID]>d_histo[up] and d_histo[channelID]>d_histo[up2] and d_histo[channelID]>d_histo[down] and d_histo[channelID]>d_histo[down2] and d_histo[channelID]>d_histo[dx] and d_histo[channelID]>d_histo[dx2])
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);
	}

	else if(threadIdx.x == threads_side and threadIdx.y>1 and threadIdx.y<threads_side-2)
	{
		if(d_histo[channelID]>threshold and d_histo[channelID]>d_histo[up] and d_histo[channelID]>d_histo[up2] and d_histo[channelID]>d_histo[down] and d_histo[channelID]>d_histo[down2] and d_histo[channelID]>d_histo[sx] and d_histo[channelID]>d_histo[sx2])
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);			
	}


	else if(threadIdx.y == 0 and threadIdx.x>1 and threadIdx.x<threads_side-2)
	{
		if(d_histo[channelID]>threshold and d_histo[channelID]>d_histo[up] and d_histo[channelID]>d_histo[up2] and d_histo[channelID]>d_histo[sx] and d_histo[channelID]>d_histo[sx2] and d_histo[channelID]>d_histo[dx] and d_histo[channelID]>d_histo[dx2])
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);
	}

	else if(threadIdx.y ==threads_side and threadIdx.x>1 and threadIdx.x<threads_side)
	{
		if(d_histo[channelID]>threshold and d_histo[channelID]>d_histo[down] and d_histo[channelID]>d_histo[down2] and d_histo[channelID]>d_histo[dx] and d_histo[channelID]>d_histo[dx2] and d_histo[channelID]>d_histo[sx] and d_histo[sx2])
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);
	}
	
	else if(threadIdx.x==1 and threadIdx.y>1 and threadIdx.y<threads_side-2)
	{
		if(d_histo[channelID] > threshold and d_histo[channelID]>=d_histo[sx] and d_histo[channelID]>=d_histo[up] and d_histo[channelID]>=d_histo[down] and d_histo[channelID]>=d_histo[down] and d_histo[channelID]>d_histo[up2] and d_histo[channelID]>d_histo[down2] and d_histo[channelID]>d_histo[dx2] and d_histo[channelID]>d_histo[ud] and d_histo[channelID]>d_histo[us] and d_histo[channelID]>d_histo[dd] and d_histo[channelID]>d_histo[ds])
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);
	}
	
	else if(threadIdx.x == threads_side-1 and threadIdx.y>1 and threadIdx.y<threads_side-2)
		if(d_histo[channelID]>threshold and d_histo[channelID]>=d_histo[up] and d_histo[channelID]>=d_histo[down] and d_histo[channelID]>=d_histo[sx]  and d_histo[channelID]>=d_histo[dx] and d_histo[channelID]>d_histo[up2] and d_histo[channelID]>d_histo[down2] and d_histo[channelID]>d_histo[sx2] and d_histo[channelID]>d_histo[ud] and d_histo[channelID]>d_histo[us] and d_histo[channelID]>d_histo[dd] and d_histo[channelID]>d_histo[ds])
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);			
	
	else if(threadIdx.y==1 and threadIdx.x>1 and threadIdx.y<threads_side-2)
		if(d_histo[channelID]>threshold and d_histo[channelID]>=d_histo[up] and d_histo[channelID]>=d_histo[dx] and d_histo[channelID]>=d_histo[sx] and d_histo[channelID]>=d_histo[down] and d_histo[channelID]>d_histo[up2] and d_histo[channelID]>d_histo[dx2] and d_histo[channelID]>d_histo[sx2] and d_histo[channelID]>d_histo[ud] and d_histo[channelID]>d_histo[us] and d_histo[channelID]>d_histo[dd] and d_histo[channelID]>d_histo[ds])
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);
				
	else if(threadIdx.y==threads_side-1 and threadIdx.x>1 and threadIdx.y<threads_side-2)
		if(d_histo[channelID]>threshold and d_histo[channelID]>=d_histo[up] and d_histo[channelID]>=d_histo[dx] and d_histo[channelID]>=d_histo[sx] and d_histo[channelID]>=d_histo[down] and d_histo[channelID]>d_histo[dx2] and d_histo[channelID]>d_histo[sx2] and d_histo[channelID]>d_histo[down2] and 
d_histo[channelID]>d_histo[ud] and d_histo[channelID]>d_histo[us] and d_histo[channelID]>d_histo[dd] and d_histo[channelID]>d_histo[ds])	
				printf("Evento %d \tMassimo %d \tCoeffAng %.3f \toffset %.3f\n", event_ID, d_histo[channelID], d_vec_coeff[bin_b], d_vec_bin[bin_a]);			
	
	
	else
		if(d_histo[channelID]>threshold)
			printf("Ciaone mi hai mancato\n");
	
}	



__global__ void histo_fill_gpu(matrix d_m, int num_bins_a, int size_xy, unsigned int *histo, float *d_vec_bin, float *d_vec_coeff)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	for(int i=0; i<size_xy; i++)
	{
		for(int j=0; j<num_bins_a; j++)
		{
			if(d_m.e[i*d_m.w+index] < d_vec_coeff[j])
			{
				atomicAdd(&histo[j*num_bins_a + index], 1);
				break;
			}
		}
	
	}
}

__global__ void prova_fill(matrix d_m, unsigned int *histo, float *d_vec_bin, float *d_vec_coeff)
{
	// Per ogni hit (totale size_xy eventi) uso un block di 1024 thread
	// per ciascun bin offset (max 1024 bin) uso una thread
	// ciascuna thread compara un canale di d_m e riempie frequenza istogramma
	int hit_id = blockIdx.x;
	int bin_offset=threadIdx.x;
	for(int bin_coeff=0; bin_coeff<num_bins_a; bin_coeff++)
	{
		if(d_m.e[hit_id*d_m.w + bin_offset] < d_vec_coeff[bin_coeff])
		{
			atomicAdd(&histo[bin_coeff*num_bins_b + bin_offset], 1);
			break;
		}

	}
}


int main()
{
	// LETTURA FILE DI CONFIGURAZIONE 
	std::string nomi_param;
	std::vector<float> parametri;
	std::ifstream infile("config.txt");
	float par_value;
	while(infile >> nomi_param >> par_value)
	{
		parametri.push_back(par_value);
	}
	infile.close();
	

// INSERISCO PARAMETRI DEL RIVELATORE E DI GENERAZIONE SEGNALI
	int num_piani= int(parametri[0]);
	float dist_interazione = parametri[1];
	float dist_piani = parametri[2];
	float dimensioni_tracker = parametri[3];
	float dim_pixel = parametri[4];
	float max_offset = parametri[10];
	int num_pixel = int((dimensioni_tracker*1000)/dim_pixel);
	int threshold = int(parametri[11]);

	float max_coeff_ang = 3;

	// INIZIO LETTURA DI FILE DATA
	// PARTO LEGGENDO FULL HEADER
	std::ifstream data("file.dat", std::ios::binary);
	full_header(data);

	// RIEMPIO UN VETTORE I CUI ELEMENTI SONO I BIN DELL'ISTOGRAMMA IN CUI VIENE SUDDIVISO L'OFFSET
	float *vettore_di_bin, *vettore_coeff_ang;
	vettore_di_bin = ( float*) malloc(num_bins_b*sizeof(float) );
	vettore_coeff_ang = (float*) malloc(num_bins_a*sizeof(float) );
	for(int i=0; i<num_bins_b; i++)
	{
		vettore_di_bin[i] = -max_offset + i*(2*max_offset/num_bins_b);
		//std::cout << vettore_di_bin[i] << " " ;
	}
	std::cout << std::endl;
	for(int j=0; j<num_bins_a; j++)
	{
		vettore_coeff_ang[j] = -max_coeff_ang + j*(2*max_coeff_ang/num_bins_a);
		//std::cout << vettore_coeff_ang[j] << " ";
	}
	std::cout << std::endl;
	// INIZIO LETTURA FRAMMENTI DATA
	while(!data.eof())
	{
		uint32_t word1;
		data.read((char*) &word1, sizeof(word1));
		uint32_t w1 = uint32_t(word1);
		std::vector<event> vec_eventi;
		if(w1==0xdeadcafe)
		{
		// CREAZIONE ISTOGRAMMA 2D: SU UN ASSE METTO IL BIN CORRIPONDENTE ALLA SUDDIVISIONE DELL'OFFSET
		// SUL SECONDO ASSE INSERISCO I COEFFICIENTI ANGOLARI TROVATI
			// LEGGO IL PAYLOAD E RIEMPIO UN VETTORE EVENT(EVENT_ID, LAYER, CHANNEL_ID)
			fragment(data, dimensioni_tracker, num_pixel, vec_eventi);
			const int size_xy = vec_eventi.size();
			uint32_t event_ID = vec_eventi[1].event_ID();

			std::cout << "frammento ricostruito correttamente" << std::endl;
			// INIZIALIZZO E RIEMPIO VETTORI DA MANDARE A GPU E CPU
			int size = size_xy*sizeof(int);
			float *vec_x, *vec_y;
			vec_x = (float *)malloc(size);
			vec_y = (float *)malloc(size);


			// PER CIASCUN EVENTO ESTRAGGO LE COORDINATE (X, Z)
			coord(vec_x, vec_y, size_xy, vec_eventi, num_pixel, dimensioni_tracker, dist_interazione, dist_piani);

			// CREO MATRICE I CUI ELEMENTI SONO I COEFFICIENTI ANGOLARI DELLE RETTE
			// CHE CONNETTONO CIASCUN BIN ALLE COORDINATE (X,Y) DELL'HIT
			matrix m;
			m.w = num_bins_b;
			m.h = size_xy;
			m.e = (float*) malloc(m.w*m.h*sizeof(float));

			// ### RIEMPIMENTO MATRICE USANDO CPU ###
			//hough_cpu(m, vettore_di_bin, vec_x, vec_y);

			// ### RIEMPIMENTO MATRICE USANDO GPU ### 
			// alloco spazio in memoria per matrice d_m
			matrix d_m;
			d_m.w = m.w;
			d_m.h = m.h;
			size_t size_matrix = d_m.w * d_m.h * sizeof(float);
			hipError_t err = hipMalloc(&d_m.e, size_matrix);
			hipError_t err1 = hipMemcpy(d_m.e, m.e, size_matrix, hipMemcpyHostToDevice);
			
			hipEvent_t start_hough, stop_hough, start_histo, stop_histo, start_max, stop_max;
			hipEventCreate(&start_hough);
			hipEventCreate(&stop_hough);
			hipEventCreate(&start_histo);
			hipEventCreate(&stop_histo);
			hipEventCreate(&start_max);
			hipEventCreate(&stop_max);

			float *d_vec_x, *d_vec_y, *d_vec_bin; 

			// Copy Coordinate Vectors from Host do Device
			size_t fragment_size = size_xy*sizeof(float);
			size_t bins_size = num_bins_b*sizeof(float);
			hipMalloc( (float **)&d_vec_x, fragment_size);
			hipMalloc( (float **)&d_vec_y, fragment_size);
			hipMalloc( (float **)&d_vec_bin, bins_size);
			hipMemcpy(d_vec_x, vec_x, fragment_size, hipMemcpyHostToDevice);
			hipMemcpy(d_vec_y, vec_y, fragment_size, hipMemcpyHostToDevice); 
			hipMemcpy(d_vec_bin, vettore_di_bin, bins_size, hipMemcpyHostToDevice);
			//partenza misura tempo 
			hipEventRecord(start_hough, 0);
			// launch hough algorith
			//hough_gpu<<<num_bins_b/64,64>>>(d_m, d_vec_bin, d_vec_x, d_vec_y, size_xy);
			//hough_gpu<<<1, num_bins_b>>>(d_m, d_vec_bin, d_vec_x, d_vec_y, size_xy);
			hough_gpu_new<<<size_xy, num_bins_b>>>(d_m, d_vec_bin, d_vec_x, d_vec_y, size_xy);
			hipError_t KernelError=hipGetLastError();
			hipDeviceSynchronize();
			float t_hough;
			hipEventRecord(stop_hough);
			hipEventSynchronize(stop_hough);

			hipEventElapsedTime(&t_hough, start_hough, stop_hough);
			// Copy matrix back to m
			hipMemcpy(m.e, d_m.e, size_matrix, hipMemcpyDeviceToHost);


			hipFree(d_vec_x);
			hipFree(d_vec_y);
			// PREPRARO ARRAY BIDIMENSIONALE IL CUI CONTENUTO SONO LE FREQUENZE DELL'ISTOGRAMMA
			
			//histo_fill_cpu(istogramma,  array_a, size_xy, m, vettore_di_bin);
			
			// Allocazione spazio memoria vettore di bin coefficiete angolare
			float *d_vec_coeff;
			gpuErrchk(hipMalloc((float**)&d_vec_coeff, num_bins_a*sizeof(float)));
			gpuErrchk(hipMemcpy(d_vec_coeff, vettore_coeff_ang,num_bins_a*sizeof(float), hipMemcpyHostToDevice));
			// Allocazione spazio di memoria per istogramma
			unsigned int *d_histo;
			unsigned int *histo;
			histo = (unsigned int*) malloc(num_bins_a*num_bins_b*sizeof(unsigned int) ); 
			gpuErrchk(hipMalloc((unsigned int**)&d_histo, num_bins_a*num_bins_b*sizeof(unsigned int)));
			hipEventRecord(start_histo);

			const int num_hits=size_xy;
			dim3 dimGrid(num_hits,1,1);
			dim3 dimBlock(num_bins_b,1, 1);
			prova_fill<<<dimGrid,dimBlock>>>(d_m, d_histo, d_vec_bin, d_vec_coeff);
			//  Riempimento istogramma  usando GPU
			//histo_fill_gpu<<<num_bins_b/128, 128>>>(d_m, num_bins_a, size_xy, d_histo, d_vec_bin, d_vec_coeff);
			hipDeviceSynchronize();
			hipEventRecord(stop_histo);
			float t_histo;
			hipEventSynchronize(stop_histo);
			hipEventElapsedTime(&t_histo, start_histo, stop_histo);

			gpuErrchk(hipMemcpy(histo, d_histo, num_bins_a*num_bins_b*sizeof(unsigned int), hipMemcpyDeviceToHost));
			hipFree(d_m.e);
			
			hipEventRecord(start_max);			
			//ricerca_max_gpu<<<num_bins_a/(num_bins_a/2),(num_bins_a/2) >>>(d_histo, d_vec_bin, d_vec_coeff, event_ID, threshold);

			int lato_a_grid = num_bins_a/threads_side;
			int lato_b_grid = num_bins_b/threads_side;
			dim3 dimBlock_histo(threads_side, threads_side, 1);
			dim3 dimGrid_histo(lato_a_grid, lato_b_grid,1);
			ricerca_max_gpu_locale2<<<dimGrid_histo, dimBlock_histo>>>(d_histo, d_vec_bin, d_vec_coeff, event_ID, threshold);

			hipDeviceSynchronize();

			hipFree(d_vec_bin);
			hipFree(d_histo);
			hipFree(d_vec_coeff);
			//hipFree(event_ID);
			//hipFree(threshold); 
			
			hipEventRecord(stop_max);
			float t_max;
			hipEventSynchronize(stop_max);
			hipEventElapsedTime(&t_max, start_max, stop_max);
			printf("Time Hough: %.6f ms\n", t_hough);
			printf("Time Histo: %.6f ms\n", t_histo);
			printf("Time Max: %.6f ms\n", t_max);


			std::cout << "fine Evento" << event_ID << std::endl;
			std::cout << "Size XY = " << size_xy << std::endl;
		}



	}	


	std::cout << "finito!" << std::endl;
	return 0;
} 
